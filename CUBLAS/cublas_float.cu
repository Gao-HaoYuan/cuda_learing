#include <cstdio>
#include <vector>
#include <random>
#include <cmath>
#include <cstring>
#include <functional>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>

#define CHECK_CUDA(x) do{auto e=(x); if(e!=hipSuccess){ \
  fprintf(stderr,"CUDA error %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(e)); exit(1);} }while(0)
#define CHECK_CUBLAS(x) do{auto s=(x); if(s!=HIPBLAS_STATUS_SUCCESS){ \
  fprintf(stderr,"cuBLAS error %s:%d: %d\n",__FILE__,__LINE__,(int)s); exit(1);} }while(0)

float time_ms(std::function<void()> fn, int warm=5, int iters=20){
    hipEvent_t a,b; CHECK_CUDA(hipEventCreate(&a)); CHECK_CUDA(hipEventCreate(&b));
    for(int i=0;i<warm;++i) fn();
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventRecord(a));
    for(int i=0;i<iters;++i) fn();
    CHECK_CUDA(hipEventRecord(b)); CHECK_CUDA(hipEventSynchronize(b));
    float ms=0; CHECK_CUDA(hipEventElapsedTime(&ms,a,b));
    CHECK_CUDA(hipEventDestroy(a)); CHECK_CUDA(hipEventDestroy(b));
    return ms/iters;
}

template<typename T>
void host_fill(std::vector<T>& v, float lo=-1.f, float hi=1.f, unsigned seed=123){
    std::mt19937 rng(seed);
    std::uniform_real_distribution<float> dist(lo,hi);
    for(auto& x : v) x = static_cast<T>(dist(rng));
}

void check_close_f32(const float* d_ref, const float* d_out, size_t n,
                     double atol, double rtol, const char* tag){
    std::vector<float> r(n), o(n);
    CHECK_CUDA(hipMemcpy(r.data(), d_ref, n*sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(o.data(), d_out, n*sizeof(float), hipMemcpyDeviceToHost));
    double max_abs=0, max_rel=0, num=0, den=0;
    for(size_t i=0;i<n;++i){
        double R=r[i], O=o[i], d=fabs(O-R);
        max_abs = std::max(max_abs, d);
        double denom = fabs(R)>0 ? fabs(R) : 1.0;
        max_rel = std::max(max_rel, d/denom);
        num += (O-R)*(O-R); den += R*R;
    }
    double rel_l2 = sqrt(num)/(sqrt(den)+1e-30);
    bool ok = (max_abs<=atol) || (max_rel<=rtol);
    printf("[%s] max_abs=%.3e  max_rel=%.3e  rel_L2=%.3e  => %s\n",
           tag, max_abs, max_rel, rel_l2, ok?"OK":"MISMATCH");
}

int main(int argc, char** argv){
    bool use_tf32 = (argc>1 && std::strcmp(argv[1],"--tf32")==0);
    printf("Mode: %s\n", use_tf32 ? "TF32 (Tensor Core for FP32)" : "Pure FP32");

    // 选一组中等偏大的尺寸（列主）。可自行修改为你的常用形状
    int M=4096, N=7546, K=1896;
    int lda=M, ldb=K, ldc=M;
    size_t sizeA=(size_t)M*K, sizeB=(size_t)K*N, sizeC=(size_t)M*N;
    printf("GEMM (col-major): C[MxN] = A[MxK] * B[KxN]\nM=%d N=%d K=%d\n", M,N,K);

    // Host init (FP32)
    std::vector<float> hA(sizeA), hB(sizeB);
    host_fill(hA); host_fill(hB);

    // Device buffers
    float *dA=nullptr, *dB=nullptr, *dC_cublas=nullptr, *dD_lt=nullptr;
    CHECK_CUDA(hipMalloc(&dA, sizeof(float)*sizeA));
    CHECK_CUDA(hipMalloc(&dB, sizeof(float)*sizeB));
    CHECK_CUDA(hipMalloc(&dC_cublas, sizeof(float)*sizeC));
    CHECK_CUDA(hipMalloc(&dD_lt, sizeof(float)*sizeC));
    CHECK_CUDA(hipMemcpy(dA, hA.data(), sizeof(float)*sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, hB.data(), sizeof(float)*sizeB, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(dC_cublas, 0, sizeof(float)*sizeC));
    CHECK_CUDA(hipMemset(dD_lt, 0, sizeof(float)*sizeC));

    // Handles
    hipblasHandle_t h; CHECK_CUBLAS(hipblasCreate(&h));
    hipblasLtHandle_t lt; CHECK_CUBLAS(hipblasLtCreate(&lt));

    // ========= cuBLAS: FP32 =========
    if(use_tf32){
        CHECK_CUBLAS(hipblasSetMathMode(h, HIPBLAS_TF32_TENSOR_OP_MATH)); // TF32 on Tensor Core
    }else{
        CHECK_CUBLAS(hipblasSetMathMode(h, HIPBLAS_DEFAULT_MATH));        // Pure FP32
    }

    float alpha=1.f, beta=0.f;
    auto run_cublas = [&](){
        // Sgemm 纯 FP32 路径（当 TF32 打开时，cuBLAS 也可能内部走 TF32/TC 快速路径）
        CHECK_CUBLAS(hipblasSgemm(
            h, HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, N, K,
            &alpha,
            dA, lda,
            dB, ldb,
            &beta,
            dC_cublas, ldc));
    };

    float t_blas = time_ms(run_cublas);
    double tflops_blas = (2.0 * (double)M * N * K) / (t_blas * 1e-3) / 1e12;
    printf("[cuBLAS ]  %.3f ms  | %.2f TFLOP/s\n", t_blas, tflops_blas);

    // ========= cuBLASLt: FP32 =========
    hipblasLtMatmulDesc_t opDesc;
    if(use_tf32){
        // TF32（FP32 输入，Tensor Core 路径）
        CHECK_CUBLAS(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F));
    }else{
        // 纯 FP32
        CHECK_CUBLAS(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    }
    hipblasOperation_t ta=HIPBLAS_OP_N, tb=HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &ta, sizeof(ta)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &tb, sizeof(tb)));

    hipblasLtMatrixLayout_t aDesc,bDesc,cDesc,dDesc;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_32F, M, K, lda));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_32F, K, N, ldb));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&cDesc, HIP_R_32F, M, N, ldc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&dDesc, HIP_R_32F, M, N, ldc));

    
    //  ========= preference + heuristic ========= 
    hipblasLtMatmulPreference_t pref;
    CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&pref));
    size_t workspaceSize = 64<<20; // 64MB
    void* workspace = nullptr; 
    CHECK_CUDA(hipMalloc(&workspace, workspaceSize));
    CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));
    hipblasLtMatmulHeuristicResult_t heur;
    int returnedResults = 0;
    CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(lt, opDesc, aDesc, bDesc, cDesc, dDesc, pref, 1, &heur, &returnedResults));

    auto run_lt = [&](){
        CHECK_CUBLAS(hipblasLtMatmul(
            lt, opDesc,
            &alpha,
            dA, aDesc,
            dB, bDesc,
            &beta,
            dD_lt, cDesc,   // C（beta=0，不使用其值），这里直接复用输出缓冲
            dD_lt, dDesc,
            &heur.algo,        // algo: 用默认即可；要更快可做 heuristic 选优
            workspace, workspaceSize, 0));
    };

    float t_lt = time_ms(run_lt);
    double tflops_lt = (2.0 * (double)M * N * K) / (t_lt * 1e-3) / 1e12;
    printf("[cuBLASLt] %.3f ms  | %.2f TFLOP/s\n", t_lt, tflops_lt);

    // ========= 精度对齐（两边都是 FP32 输出） =========
    // 纯 FP32：使用严格容差；TF32：容差稍松
    double atol = use_tf32 ? 1e-4 : 1e-6;
    double rtol = use_tf32 ? 1e-3 : 1e-5;
    check_close_f32(dC_cublas, dD_lt, sizeC, atol, rtol,
                    use_tf32 ? "cuBLAS vs cuBLASLt (TF32)" : "cuBLAS vs cuBLASLt (FP32)");

    // 清理
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(aDesc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(bDesc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(cDesc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(dDesc));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(opDesc));
    CHECK_CUBLAS(hipblasLtDestroy(lt));
    CHECK_CUBLAS(hipblasDestroy(h));
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC_cublas));
    CHECK_CUDA(hipFree(dD_lt));
    return 0;
}

// 这是 test.cu文件的内容
#include <cstdio>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {
#ifdef __CUDA_ARCH__
    printf("Hello, world from GPU architecture %d!\n", __CUDA_ARCH__);
#else
    printf("Hello, world from CPU!\n");
#endif
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 2>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "minitest.hpp"

// 检查报错
#define CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl; \
        exit(1); \
    }

const int N = 256;

__global__ void initKernel(int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] = idx;  // 将每个元素初始化为其索引
    }
}

__global__ void squareKernel(int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] *= data[idx];  // 将每个元素平方
    }
}

TEST(CUDA, Graph) {
    int *d_data, *h_data;
    hipMalloc(&d_data, N * sizeof(int));
    h_data = (int*)malloc(N * sizeof(int));

    // 创建 graph
    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    // 设置通用的 kernel 参数
    dim3 block(32);
    dim3 grid((N + block.x - 1) / block.x);

    // ===== 创建第一个 kernel 节点：初始化 =====
    hipKernelNodeParams initParams = {};
    void *initArgs[] = { &d_data };
    initParams.func = (void*)initKernel;
    initParams.gridDim = grid;
    initParams.blockDim = block;
    initParams.kernelParams = initArgs;
    initParams.sharedMemBytes = 0;

    hipGraphNode_t initNode;
    hipGraphAddKernelNode(&initNode, graph, nullptr, 0, &initParams);

    // ===== 创建第二个 kernel 节点：平方 =====
    hipKernelNodeParams squareParams = {};
    void *squareArgs[] = { &d_data };
    squareParams.func = (void*)squareKernel;
    squareParams.gridDim = grid;
    squareParams.blockDim = block;
    squareParams.kernelParams = squareArgs;
    squareParams.sharedMemBytes = 0;

    hipGraphNode_t squareNode;
    // 设置依赖关系：squareNode 依赖于 initNode
    hipGraphAddKernelNode(&squareNode, graph, &initNode, 1, &squareParams);

    // 实例化并执行图
    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);

    // 拷贝结果回 host 并打印
    hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        std::cout << "h_data[" << i << "] = " << h_data[i] << std::endl;
    }

    // 清理资源
    hipFree(d_data);
    free(h_data);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(stream);
}

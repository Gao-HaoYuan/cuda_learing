#include "hip/hip_runtime.h"
// kernel.cu
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

using at::Tensor;

__global__ void curand_normal_kernel(
    float* __restrict__ out,
    int64_t N,
    unsigned long long seed,
    float mean,
    float std)
{
    int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = blockDim.x * gridDim.x;

    // 每个线程独立的 RNG 状态
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, tid, 0, &state);  // (seed, sequence, offset, state)

    for (int64_t i = tid; i < N; i += stride) {
        float r = hiprand_normal(&state); // mean=0, std=1
        out[i] = mean + std * r;
    }
}

void launch_curand_normal_kernel(at::Tensor out, unsigned long long seed, float mean, float std)
{
    TORCH_CHECK(out.is_cuda(), "output must be CUDA tensor");
    TORCH_CHECK(out.scalar_type() == at::kFloat, "only float32 supported");

    const int64_t N = out.numel();
    if (N == 0) return;

    const int threads = 256;
    const int blocks = std::min<int>((N + threads - 1) / threads, 4096);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    curand_normal_kernel<<<blocks, threads, 0, stream>>>(
        out.data_ptr<float>(), N, seed, mean, std);

    AT_CUDA_CHECK(hipGetLastError());
}
